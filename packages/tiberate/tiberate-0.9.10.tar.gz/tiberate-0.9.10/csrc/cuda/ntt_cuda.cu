#include "hip/hip_runtime.h"
#include "ntt_cuda.h"
#include <c10/cuda/CUDAStream.h>
#include "mont_cuda_kernel.cuh"

#define BLOCK_SIZE 256

//------------------------------------------------------------------
// ntt
//------------------------------------------------------------------

template <typename scalar_t>
__global__ void ntt_cuda_kernel(
    torch::PackedTensorAccessor32<scalar_t, 2> a_acc,
    const torch::PackedTensorAccessor32<int, 2> even_acc,
    const torch::PackedTensorAccessor32<int, 2> odd_acc,
    const torch::PackedTensorAccessor32<scalar_t, 3> psi_acc,
    const torch::PackedTensorAccessor32<scalar_t, 1> _2q_acc,
    const torch::PackedTensorAccessor32<scalar_t, 1> ql_acc,
    const torch::PackedTensorAccessor32<scalar_t, 1> qh_acc,
    const torch::PackedTensorAccessor32<scalar_t, 1> kl_acc,
    const torch::PackedTensorAccessor32<scalar_t, 1> kh_acc,
    const int level) {
  // Where am I?
  const int i = blockIdx.x;
  const int j = blockIdx.y * BLOCK_SIZE + threadIdx.x;

  // Montgomery inputs.
  const scalar_t _2q = _2q_acc[i];
  const scalar_t ql = ql_acc[i];
  const scalar_t qh = qh_acc[i];
  const scalar_t kl = kl_acc[i];
  const scalar_t kh = kh_acc[i];

  // Butterfly.
  const int even_j = even_acc[level][j];
  const int odd_j = odd_acc[level][j];

  const scalar_t U = a_acc[i][even_j];
  const scalar_t S = psi_acc[i][level][j];
  const scalar_t O = a_acc[i][odd_j];
  const scalar_t V = mont_mult_scalar_cuda_kernel(S, O, ql, qh, kl, kh);

  // Store back.
  const scalar_t UplusV = U + V;
  const scalar_t UminusV = U + _2q - V;

  a_acc[i][even_j] = (UplusV < _2q) ? UplusV : UplusV - _2q;
  a_acc[i][odd_j] = (UminusV < _2q) ? UminusV : UminusV - _2q;
}

template <typename scalar_t>
void ntt_cuda_typed(torch::Tensor a,
                    const torch::Tensor even,
                    const torch::Tensor odd,
                    const torch::Tensor psi,
                    const torch::Tensor _2q,
                    const torch::Tensor ql,
                    const torch::Tensor qh,
                    const torch::Tensor kl,
                    const torch::Tensor kh) {
  // Retrieve the device index, then set the corresponding device and stream.
  auto device_id = a.device().index();
  hipSetDevice(device_id);

  // Use a preallocated pytorch stream.
  auto stream = at::cuda::getCurrentCUDAStream(device_id);

  // The problem dimension.
  const auto C = ql.size(0);
  const auto logN = even.size(0);
  const auto N = even.size(1);

  int dim_block = BLOCK_SIZE;
  dim3 dim_grid(C, N / BLOCK_SIZE);

  // Run the cuda kernel.
  auto a_acc = a.packed_accessor32<scalar_t, 2>();

  const auto even_acc = even.packed_accessor32<int, 2>();
  const auto odd_acc = odd.packed_accessor32<int, 2>();
  const auto psi_acc = psi.packed_accessor32<scalar_t, 3>();

  const auto _2q_acc = _2q.packed_accessor32<scalar_t, 1>();
  const auto ql_acc = ql.packed_accessor32<scalar_t, 1>();
  const auto qh_acc = qh.packed_accessor32<scalar_t, 1>();
  const auto kl_acc = kl.packed_accessor32<scalar_t, 1>();
  const auto kh_acc = kh.packed_accessor32<scalar_t, 1>();

  for (int i = 0; i < logN; ++i) {
    ntt_cuda_kernel<scalar_t><<<dim_grid, dim_block, 0, stream>>>(a_acc,
                                                                  even_acc,
                                                                  odd_acc,
                                                                  psi_acc,
                                                                  _2q_acc,
                                                                  ql_acc,
                                                                  qh_acc,
                                                                  kl_acc,
                                                                  kh_acc,
                                                                  i);
  }
}

void ntt_cuda(torch::Tensor a,
              const torch::Tensor even,
              const torch::Tensor odd,
              const torch::Tensor psi,
              const torch::Tensor _2q,
              const torch::Tensor ql,
              const torch::Tensor qh,
              const torch::Tensor kl,
              const torch::Tensor kh) {
  // Dispatch to the correct data type.
  AT_DISPATCH_INTEGRAL_TYPES(a.scalar_type(), "typed_ntt_cuda", ([&] {
                               ntt_cuda_typed<scalar_t>(
                                   a, even, odd, psi, _2q, ql, qh, kl, kh);
                             }));
}

//------------------------------------------------------------------
// enter_ntt
//------------------------------------------------------------------

template <typename scalar_t>
void enter_ntt_cuda_typed(torch::Tensor a,
                          const torch::Tensor Rs,
                          const torch::Tensor even,
                          const torch::Tensor odd,
                          const torch::Tensor psi,
                          const torch::Tensor _2q,
                          const torch::Tensor ql,
                          const torch::Tensor qh,
                          const torch::Tensor kl,
                          const torch::Tensor kh) {
  // Retrieve the device index, then set the corresponding device and stream.
  auto device_id = a.device().index();
  hipSetDevice(device_id);

  // Use a preallocated pytorch stream.
  auto stream = at::cuda::getCurrentCUDAStream(device_id);

  // The problem dimension.
  // Be careful. even and odd has half the length of the a.
  const auto C = ql.size(0);
  const auto logN = even.size(0);
  const auto N_half = even.size(1);
  const auto N = a.size(1);

  int dim_block = BLOCK_SIZE;
  dim3 dim_grid_ntt(C, N_half / BLOCK_SIZE);
  dim3 dim_grid_enter(C, N / BLOCK_SIZE);

  // Run the cuda kernel.
  auto a_acc = a.packed_accessor32<scalar_t, 2>();
  const auto Rs_acc = Rs.packed_accessor32<scalar_t, 1>();

  const auto even_acc = even.packed_accessor32<int, 2>();
  const auto odd_acc = odd.packed_accessor32<int, 2>();
  const auto psi_acc = psi.packed_accessor32<scalar_t, 3>();

  const auto _2q_acc = _2q.packed_accessor32<scalar_t, 1>();
  const auto ql_acc = ql.packed_accessor32<scalar_t, 1>();
  const auto qh_acc = qh.packed_accessor32<scalar_t, 1>();
  const auto kl_acc = kl.packed_accessor32<scalar_t, 1>();
  const auto kh_acc = kh.packed_accessor32<scalar_t, 1>();

  // enter.
  mont_enter_cuda_kernel<scalar_t><<<dim_grid_enter, dim_block, 0, stream>>>(
      a_acc, Rs_acc, ql_acc, qh_acc, kl_acc, kh_acc);

  // ntt.
  for (int i = 0; i < logN; ++i) {
    ntt_cuda_kernel<scalar_t><<<dim_grid_ntt, dim_block, 0, stream>>>(a_acc,
                                                                      even_acc,
                                                                      odd_acc,
                                                                      psi_acc,
                                                                      _2q_acc,
                                                                      ql_acc,
                                                                      qh_acc,
                                                                      kl_acc,
                                                                      kh_acc,
                                                                      i);
  }
}

void enter_ntt_cuda(torch::Tensor a,
                    const torch::Tensor Rs,
                    const torch::Tensor even,
                    const torch::Tensor odd,
                    const torch::Tensor psi,
                    const torch::Tensor _2q,
                    const torch::Tensor ql,
                    const torch::Tensor qh,
                    const torch::Tensor kl,
                    const torch::Tensor kh) {
  // Dispatch to the correct data type.
  AT_DISPATCH_INTEGRAL_TYPES(a.scalar_type(), "typed_enter_ntt_cuda", ([&] {
                               enter_ntt_cuda_typed<scalar_t>(
                                   a, Rs, even, odd, psi, _2q, ql, qh, kl, kh);
                             }));
}

//------------------------------------------------------------------
// intt
//------------------------------------------------------------------

template <typename scalar_t>
__global__ void intt_cuda_kernel(
    torch::PackedTensorAccessor32<scalar_t, 2> a_acc,
    const torch::PackedTensorAccessor32<int, 2> even_acc,
    const torch::PackedTensorAccessor32<int, 2> odd_acc,
    const torch::PackedTensorAccessor32<scalar_t, 3> psi_acc,
    const torch::PackedTensorAccessor32<scalar_t, 1> _2q_acc,
    const torch::PackedTensorAccessor32<scalar_t, 1> ql_acc,
    const torch::PackedTensorAccessor32<scalar_t, 1> qh_acc,
    const torch::PackedTensorAccessor32<scalar_t, 1> kl_acc,
    const torch::PackedTensorAccessor32<scalar_t, 1> kh_acc,
    const int level) {
  // Where am I?
  const int i = blockIdx.x;
  const int j = blockIdx.y * BLOCK_SIZE + threadIdx.x;

  // Montgomery inputs.
  const scalar_t _2q = _2q_acc[i];
  const scalar_t ql = ql_acc[i];
  const scalar_t qh = qh_acc[i];
  const scalar_t kl = kl_acc[i];
  const scalar_t kh = kh_acc[i];

  // Butterfly.
  const int even_j = even_acc[level][j];
  const int odd_j = odd_acc[level][j];

  const scalar_t U = a_acc[i][even_j];
  const scalar_t S = psi_acc[i][level][j];
  const scalar_t V = a_acc[i][odd_j];

  const scalar_t UminusV = U + _2q - V;
  const scalar_t O = (UminusV < _2q) ? UminusV : UminusV - _2q;

  const scalar_t W = mont_mult_scalar_cuda_kernel(S, O, ql, qh, kl, kh);
  a_acc[i][odd_j] = W;

  const scalar_t UplusV = U + V;
  a_acc[i][even_j] = (UplusV < _2q) ? UplusV : UplusV - _2q;
}

template <typename scalar_t>
void intt_cuda_typed(torch::Tensor a,
                     const torch::Tensor even,
                     const torch::Tensor odd,
                     const torch::Tensor psi,
                     const torch::Tensor Ninv,
                     const torch::Tensor _2q,
                     const torch::Tensor ql,
                     const torch::Tensor qh,
                     const torch::Tensor kl,
                     const torch::Tensor kh) {
  // Retrieve the device index, then set the corresponding device and stream.
  auto device_id = a.device().index();
  hipSetDevice(device_id);

  // Use a preallocated pytorch stream.
  auto stream = at::cuda::getCurrentCUDAStream(device_id);

  // The problem dimension.
  // Be careful. even and odd has half the length of the a.
  const auto C = ql.size(0);
  const auto logN = even.size(0);
  const auto N_half = even.size(1);
  const auto N = a.size(1);

  int dim_block = BLOCK_SIZE;
  dim3 dim_grid_ntt(C, N_half / BLOCK_SIZE);
  dim3 dim_grid_enter(C, N / BLOCK_SIZE);

  // Run the cuda kernel.
  auto a_acc = a.packed_accessor32<scalar_t, 2>();

  const auto even_acc = even.packed_accessor32<int, 2>();
  const auto odd_acc = odd.packed_accessor32<int, 2>();
  const auto psi_acc = psi.packed_accessor32<scalar_t, 3>();
  const auto Ninv_acc = Ninv.packed_accessor32<scalar_t, 1>();

  const auto _2q_acc = _2q.packed_accessor32<scalar_t, 1>();
  const auto ql_acc = ql.packed_accessor32<scalar_t, 1>();
  const auto qh_acc = qh.packed_accessor32<scalar_t, 1>();
  const auto kl_acc = kl.packed_accessor32<scalar_t, 1>();
  const auto kh_acc = kh.packed_accessor32<scalar_t, 1>();

  for (int i = 0; i < logN; ++i) {
    intt_cuda_kernel<scalar_t><<<dim_grid_ntt, dim_block, 0, stream>>>(a_acc,
                                                                       even_acc,
                                                                       odd_acc,
                                                                       psi_acc,
                                                                       _2q_acc,
                                                                       ql_acc,
                                                                       qh_acc,
                                                                       kl_acc,
                                                                       kh_acc,
                                                                       i);
  }

  // Normalize.
  mont_enter_cuda_kernel<scalar_t><<<dim_grid_enter, dim_block, 0, stream>>>(
      a_acc, Ninv_acc, ql_acc, qh_acc, kl_acc, kh_acc);
}

void intt_cuda(torch::Tensor a,
               const torch::Tensor even,
               const torch::Tensor odd,
               const torch::Tensor psi,
               const torch::Tensor Ninv,
               const torch::Tensor _2q,
               const torch::Tensor ql,
               const torch::Tensor qh,
               const torch::Tensor kl,
               const torch::Tensor kh) {
  // Dispatch to the correct data type.
  AT_DISPATCH_INTEGRAL_TYPES(
      a.scalar_type(), "typed_intt_cuda", ([&] {
        intt_cuda_typed<scalar_t>(a, even, odd, psi, Ninv, _2q, ql, qh, kl, kh);
      }));
}

//------------------------------------------------------------------
// Chained intt series.
//------------------------------------------------------------------
// intt exit

template <typename scalar_t>
void intt_exit_cuda_typed(torch::Tensor a,
                          const torch::Tensor even,
                          const torch::Tensor odd,
                          const torch::Tensor psi,
                          const torch::Tensor Ninv,
                          const torch::Tensor _2q,
                          const torch::Tensor ql,
                          const torch::Tensor qh,
                          const torch::Tensor kl,
                          const torch::Tensor kh) {
  // Retrieve the device index, then set the corresponding device and stream.
  auto device_id = a.device().index();
  hipSetDevice(device_id);

  // Use a preallocated pytorch stream.
  auto stream = at::cuda::getCurrentCUDAStream(device_id);

  // The problem dimension.
  // Be careful. even and odd has half the length of the a.
  const auto C = ql.size(0);
  const auto logN = even.size(0);
  const auto N_half = even.size(1);
  const auto N = a.size(1);

  int dim_block = BLOCK_SIZE;
  dim3 dim_grid_ntt(C, N_half / BLOCK_SIZE);
  dim3 dim_grid_enter(C, N / BLOCK_SIZE);

  // Run the cuda kernel.
  auto a_acc = a.packed_accessor32<scalar_t, 2>();

  const auto even_acc = even.packed_accessor32<int, 2>();
  const auto odd_acc = odd.packed_accessor32<int, 2>();
  const auto psi_acc = psi.packed_accessor32<scalar_t, 3>();
  const auto Ninv_acc = Ninv.packed_accessor32<scalar_t, 1>();

  const auto _2q_acc = _2q.packed_accessor32<scalar_t, 1>();
  const auto ql_acc = ql.packed_accessor32<scalar_t, 1>();
  const auto qh_acc = qh.packed_accessor32<scalar_t, 1>();
  const auto kl_acc = kl.packed_accessor32<scalar_t, 1>();
  const auto kh_acc = kh.packed_accessor32<scalar_t, 1>();

  for (int i = 0; i < logN; ++i) {
    intt_cuda_kernel<scalar_t><<<dim_grid_ntt, dim_block, 0, stream>>>(a_acc,
                                                                       even_acc,
                                                                       odd_acc,
                                                                       psi_acc,
                                                                       _2q_acc,
                                                                       ql_acc,
                                                                       qh_acc,
                                                                       kl_acc,
                                                                       kh_acc,
                                                                       i);
  }

  // Normalize.
  mont_enter_cuda_kernel<scalar_t><<<dim_grid_enter, dim_block, 0, stream>>>(
      a_acc, Ninv_acc, ql_acc, qh_acc, kl_acc, kh_acc);

  // Exit.
  mont_reduce_cuda_kernel<scalar_t><<<dim_grid_enter, dim_block, 0, stream>>>(
      a_acc, ql_acc, qh_acc, kl_acc, kh_acc);
}

// intt exit reduce

template <typename scalar_t>
void intt_exit_reduce_cuda_typed(torch::Tensor a,
                                 const torch::Tensor even,
                                 const torch::Tensor odd,
                                 const torch::Tensor psi,
                                 const torch::Tensor Ninv,
                                 const torch::Tensor _2q,
                                 const torch::Tensor ql,
                                 const torch::Tensor qh,
                                 const torch::Tensor kl,
                                 const torch::Tensor kh) {
  // Retrieve the device index, then set the corresponding device and stream.
  auto device_id = a.device().index();
  hipSetDevice(device_id);

  // Use a preallocated pytorch stream.
  auto stream = at::cuda::getCurrentCUDAStream(device_id);

  // The problem dimension.
  // Be careful. even and odd has half the length of the a.
  const auto C = ql.size(0);
  const auto logN = even.size(0);
  const auto N_half = even.size(1);
  const auto N = a.size(1);

  int dim_block = BLOCK_SIZE;
  dim3 dim_grid_ntt(C, N_half / BLOCK_SIZE);
  dim3 dim_grid_enter(C, N / BLOCK_SIZE);

  // Run the cuda kernel.
  auto a_acc = a.packed_accessor32<scalar_t, 2>();

  const auto even_acc = even.packed_accessor32<int, 2>();
  const auto odd_acc = odd.packed_accessor32<int, 2>();
  const auto psi_acc = psi.packed_accessor32<scalar_t, 3>();
  const auto Ninv_acc = Ninv.packed_accessor32<scalar_t, 1>();

  const auto _2q_acc = _2q.packed_accessor32<scalar_t, 1>();
  const auto ql_acc = ql.packed_accessor32<scalar_t, 1>();
  const auto qh_acc = qh.packed_accessor32<scalar_t, 1>();
  const auto kl_acc = kl.packed_accessor32<scalar_t, 1>();
  const auto kh_acc = kh.packed_accessor32<scalar_t, 1>();

  for (int i = 0; i < logN; ++i) {
    intt_cuda_kernel<scalar_t><<<dim_grid_ntt, dim_block, 0, stream>>>(a_acc,
                                                                       even_acc,
                                                                       odd_acc,
                                                                       psi_acc,
                                                                       _2q_acc,
                                                                       ql_acc,
                                                                       qh_acc,
                                                                       kl_acc,
                                                                       kh_acc,
                                                                       i);
  }

  // Normalize.
  mont_enter_cuda_kernel<scalar_t><<<dim_grid_enter, dim_block, 0, stream>>>(
      a_acc, Ninv_acc, ql_acc, qh_acc, kl_acc, kh_acc);

  // Exit.
  mont_reduce_cuda_kernel<scalar_t><<<dim_grid_enter, dim_block, 0, stream>>>(
      a_acc, ql_acc, qh_acc, kl_acc, kh_acc);

  // Reduce.
  reduce_2q_cuda_kernel<scalar_t>
      <<<dim_grid_enter, dim_block, 0, stream>>>(a_acc, _2q_acc);
}

///////////////////////////////////////////////////////////////
// intt exit reduce signed

template <typename scalar_t>
void intt_exit_reduce_signed_cuda_typed(torch::Tensor a,
                                        const torch::Tensor even,
                                        const torch::Tensor odd,
                                        const torch::Tensor psi,
                                        const torch::Tensor Ninv,
                                        const torch::Tensor _2q,
                                        const torch::Tensor ql,
                                        const torch::Tensor qh,
                                        const torch::Tensor kl,
                                        const torch::Tensor kh) {
  // Retrieve the device index, then set the corresponding device and stream.
  auto device_id = a.device().index();
  hipSetDevice(device_id);

  // Use a preallocated pytorch stream.
  auto stream = at::cuda::getCurrentCUDAStream(device_id);

  // The problem dimension.
  // Be careful. even and odd has half the length of the a.
  const auto C = ql.size(0);
  const auto logN = even.size(0);
  const auto N_half = even.size(1);
  const auto N = a.size(1);

  int dim_block = BLOCK_SIZE;
  dim3 dim_grid_ntt(C, N_half / BLOCK_SIZE);
  dim3 dim_grid_enter(C, N / BLOCK_SIZE);

  // Run the cuda kernel.
  auto a_acc = a.packed_accessor32<scalar_t, 2>();

  const auto even_acc = even.packed_accessor32<int, 2>();
  const auto odd_acc = odd.packed_accessor32<int, 2>();
  const auto psi_acc = psi.packed_accessor32<scalar_t, 3>();
  const auto Ninv_acc = Ninv.packed_accessor32<scalar_t, 1>();

  const auto _2q_acc = _2q.packed_accessor32<scalar_t, 1>();
  const auto ql_acc = ql.packed_accessor32<scalar_t, 1>();
  const auto qh_acc = qh.packed_accessor32<scalar_t, 1>();
  const auto kl_acc = kl.packed_accessor32<scalar_t, 1>();
  const auto kh_acc = kh.packed_accessor32<scalar_t, 1>();

  for (int i = 0; i < logN; ++i) {
    intt_cuda_kernel<scalar_t><<<dim_grid_ntt, dim_block, 0, stream>>>(a_acc,
                                                                       even_acc,
                                                                       odd_acc,
                                                                       psi_acc,
                                                                       _2q_acc,
                                                                       ql_acc,
                                                                       qh_acc,
                                                                       kl_acc,
                                                                       kh_acc,
                                                                       i);
  }

  // Normalize.
  mont_enter_cuda_kernel<scalar_t><<<dim_grid_enter, dim_block, 0, stream>>>(
      a_acc, Ninv_acc, ql_acc, qh_acc, kl_acc, kh_acc);

  // Exit.
  mont_reduce_cuda_kernel<scalar_t><<<dim_grid_enter, dim_block, 0, stream>>>(
      a_acc, ql_acc, qh_acc, kl_acc, kh_acc);

  // Reduce.
  reduce_2q_cuda_kernel<scalar_t>
      <<<dim_grid_enter, dim_block, 0, stream>>>(a_acc, _2q_acc);

  // Make signed.
  make_signed_cuda_kernel<scalar_t>
      <<<dim_grid_enter, dim_block, 0, stream>>>(a_acc, _2q_acc);
}

/**************************************************************/
/* Connectors                                                 */
/**************************************************************/

///////////////////////////////////////////////////////////////
// intt exit

void intt_exit_cuda(torch::Tensor a,
                    const torch::Tensor even,
                    const torch::Tensor odd,
                    const torch::Tensor psi,
                    const torch::Tensor Ninv,
                    const torch::Tensor _2q,
                    const torch::Tensor ql,
                    const torch::Tensor qh,
                    const torch::Tensor kl,
                    const torch::Tensor kh) {
  // Dispatch to the correct data type.
  AT_DISPATCH_INTEGRAL_TYPES(
      a.scalar_type(), "typed_intt_exit_cuda", ([&] {
        intt_exit_cuda_typed<scalar_t>(
            a, even, odd, psi, Ninv, _2q, ql, qh, kl, kh);
      }));
}

///////////////////////////////////////////////////////////////
// intt exit reduce

void intt_exit_reduce_cuda(torch::Tensor a,
                           const torch::Tensor even,
                           const torch::Tensor odd,
                           const torch::Tensor psi,
                           const torch::Tensor Ninv,
                           const torch::Tensor _2q,
                           const torch::Tensor ql,
                           const torch::Tensor qh,
                           const torch::Tensor kl,
                           const torch::Tensor kh) {
  // Dispatch to the correct data type.
  AT_DISPATCH_INTEGRAL_TYPES(
      a.scalar_type(), "typed_intt_exit_reduce_cuda", ([&] {
        intt_exit_reduce_cuda_typed<scalar_t>(
            a, even, odd, psi, Ninv, _2q, ql, qh, kl, kh);
      }));
}

// intt exit reduce signed

void intt_exit_reduce_signed_cuda(torch::Tensor a,
                                  const torch::Tensor even,
                                  const torch::Tensor odd,
                                  const torch::Tensor psi,
                                  const torch::Tensor Ninv,
                                  const torch::Tensor _2q,
                                  const torch::Tensor ql,
                                  const torch::Tensor qh,
                                  const torch::Tensor kl,
                                  const torch::Tensor kh) {
  // Dispatch to the correct data type.
  AT_DISPATCH_INTEGRAL_TYPES(
      a.scalar_type(), "typed_intt_exit_reduce_signed_cuda", ([&] {
        intt_exit_reduce_signed_cuda_typed<scalar_t>(
            a, even, odd, psi, Ninv, _2q, ql, qh, kl, kh);
      }));
}
